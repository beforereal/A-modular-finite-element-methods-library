/*******************************************************************************
To compile: gcc -O3 -o mandelbrot mandelbrot.c -lm
To create an image with 4096 x 4096 pixels: ./mandelbrot 4096 4096 
*******************************************************************************/
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"



void writeMandelbrot(const char *fileName, int width, int height, float *img, int minI, int maxI);

#define MXITER 1000

/*******************************************************************************/
// Define a complex number
typedef struct {
  double x;
  double y;
}complex_t;


/*******************************************************************************/
// Return iterations before z leaves mandelbrot set for given c
__device__ int testpoint(complex_t c){
  int iter;  
  complex_t z = c;

  for(iter=0; iter<MXITER; iter++){ 
    // real part of z^2 + c 
    double tmp = (z.x*z.x) - (z.y*z.y) + c.x;
    // update with imaginary part of z^2 + c
    z.y = z.x*z.y*2. + c.y;
    // update real part
    z.x = tmp; 
    // check bound
    if((z.x*z.x+z.y*z.y)>4.0){ return iter;}
  }
  return iter; 
}

/*******************************************************************************/
// perform Mandelbrot iteration on a grid of numbers in the complex plane
// record the  iteration counts in the count array
__global__ void mandelbrotKernel(int Nre, int Nim, complex_t cmin, complex_t dc, float *count) {
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    if (m < Nre && n < Nim) {
        complex_t c;
        c.x = cmin.x + dc.x * m;
        c.y = cmin.y + dc.y * n;
        count[m + n * Nre] = (float)testpoint(c);
    }
}

/*******************************************************************************/

void computeMandelbrotGPU(int Nre, int Nim, complex_t cmin, complex_t dc, float *count) {
    dim3 blockSize(16, 16);  // Adjust block size as needed
    dim3 gridSize((Nre + blockSize.x - 1) / blockSize.x, (Nim + blockSize.y - 1) / blockSize.y);

    mandelbrotKernel<<<gridSize, blockSize>>>(Nre, Nim, cmin, dc, count);
    hipDeviceSynchronize();
}



/*******************************************************************************/




int main(int argc, char **argv){

  // to create a 4096x4096 pixel image 
  // usage: ./mandelbrot 4096 4096 
  
  int Nre = (argc==3) ? atoi(argv[1]): 4096;
  int Nim = (argc==3) ? atoi(argv[2]): 4096;
  
  // storage for the iteration counts
  float *count;
  count = (float*) malloc(Nre*Nim*sizeof(float));
      float *count_d;
    hipMalloc((void**)&count_d, Nre * Nim * sizeof(float));

  // Parameters for a bounding box for "c" that generates an interesting image
  // const float centRe = -.759856, centIm= .125547;
  // const float diam  = 0.151579;
  const float centRe = -0.5, centIm= 0;
  const float diam  = 3.0;

  complex_t cmin; 
  complex_t cmax;
  complex_t dc;

  cmin.x = centRe - 0.5*diam;
  cmax.x = centRe + 0.5*diam;
  cmin.y = centIm - 0.5*diam;
  cmax.y = centIm + 0.5*diam;

  //set step sizes
dc.x = (cmax.x-cmin.x)/(Nre-1);
dc.y = (cmax.y-cmin.y)/(Nim-1);

hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

// Ensure device synchronization before measuring time
hipDeviceSynchronize();
hipEventRecord(start, 0);

// compute mandelbrot set
computeMandelbrotGPU(Nre, Nim, cmin, dc, count_d);
  
// copy from the GPU back to the host here
hipMemcpy(count, count_d, Nre * Nim * sizeof(float), hipMemcpyDeviceToHost);
  
// Ensure device synchronization before measuring elapsed time
hipDeviceSynchronize();
hipEventRecord(stop, 0);
hipEventSynchronize(stop);

float elapsedTime;
hipEventElapsedTime(&elapsedTime, start, stop);
printf("Elapsed time (CUDA): %f ms\n", elapsedTime);

  // output mandelbrot to ppm format image
  printf("Printing mandelbrot.ppm...");
  writeMandelbrot("mandelbrot.ppm", Nre, Nim, count, 0, 80);
  printf("done.\n");

  hipFree(count_d);	
  free(count);

  exit(0);
  return 0;
}  


/* Output data as PPM file */
void saveppm(const char *filename, unsigned char *img, int width, int height){

  /* FILE pointer */
  FILE *f;
  
  /* Open file for writing */
  f = fopen(filename, "wb");
  
  /* PPM header info, including the size of the image */
  fprintf(f, "P6 %d %d %d\n", width, height, 255);

  /* Write the image data to the file - remember 3 byte per pixel */
  fwrite(img, 3, width*height, f);

  /* Make sure you close the file */
  fclose(f);
}



void writeMandelbrot(const char *fileName, int width, int height, float *img, int minI, int maxI){

  int n, m;
  unsigned char *rgb   = (unsigned char*) calloc(3*width*height, sizeof(unsigned char));
  
  for(n=0;n<height;++n){
    for(m=0;m<width;++m){
      int id = m+n*width;
      int I = (int) (768*sqrt((double)(img[id]-minI)/(maxI-minI)));
      
      // change this to change palette
      if(I<256)      rgb[3*id+2] = 255-I;
      else if(I<512) rgb[3*id+1] = 511-I;
      else if(I<768) rgb[3*id+0] = 767-I;
      else if(I<1024) rgb[3*id+0] = 1023-I;
      else if(I<1536) rgb[3*id+1] = 1535-I;
      else if(I<2048) rgb[3*id+2] = 2047-I;

    }
  }

  saveppm(fileName, rgb, width, height);

  free(rgb);
}
